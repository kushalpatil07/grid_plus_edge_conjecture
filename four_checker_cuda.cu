
#include <hip/hip_runtime.h>
#include <math.h>

#include <iostream>
#include <set>
#include <string>
#include <tuple>
#include <utility>
#include <vector>

#define TPB 256

using namespace std;

__host__ __device__ int dist(int x1, int y1, int x2, int y2) {
    return abs(x1 - x2) + abs(y1 - y2);
}

__device__ int distf(int x1, int y1, int x2, int y2, int x3, int y3, int x4,
                     int y4) {
    return min(dist(x1, y1, x2, y2),
               min(dist(x1, y1, x3, y3) + dist(x4, y4, x2, y2) + 1,
                   dist(x1, y1, x4, y4) + dist(x3, y3, x2, y2) + 1));
}

__device__ bool corner_check(int x, int y, int n, int m) {
    if (x == 1) {
        if (y == 1 || y == m) return false;
    } else if (x == n) {
        if (y == m || y == 1) return false;
    }
    return true;
}

__global__ void four_cond(int* x1, int* y1, int* x2, int* y2, int n, int m,
                          int num_pairs, bool* out) {
    int idx = blockIdx.x * TPB + threadIdx.x;
    if (idx >= num_pairs) return;
    int Gain1 = abs(abs(y1[idx] - y2[idx]) - abs(x1[idx] - x2[idx])) - 1;
    bool first = corner_check(x1[idx], y1[idx], n, m) &&
                 corner_check(x2[idx], y2[idx], n, m);
    bool second = (Gain1 % 2 == 0) && (Gain1 > 0);
    bool third =
        2 * min(abs(x2[idx] - x1[idx]), abs(y1[idx] - y2[idx])) >= (Gain1 + 4);
    out[idx] = first && second && third;
    return;
}

__global__ void make_three_points(int* px, int* py, int* tpx1, int* tpy1,
                                  int* tpx2, int* tpy2, int* tpx3, int* tpy3,
                                  int n, int m) {
    int i = blockIdx.x * TPB + threadIdx.x;
    int ntp = (n * m * (n * m - 1) * (n * m - 2)) / 6;
    if (i >= ntp) return;
    int c = 0;
    for (int i = 0; i < n * m; i++) {
        for (int j = i + 1; j < n * m; j++) {
            for (int k = j + 1; k < n * m; k++) {
                tpx1[c] = px[i];
                tpy1[c] = py[i];
                tpx2[c] = px[j];
                tpy2[c] = py[j];
                tpx3[c] = px[k];
                tpy3[c] = py[k];
                c++;
            }
        }
    }
    return;
}

__global__ void find(int* ppx1, int* ppy1, int* ppx2, int* ppy2, int* tpx1,
                     int* tpy1, int* tpx2, int* tpy2, int* tpx3, int* tpy3,
                     int* px, int* py, int n, int m, int npp, bool* out) {
    int i = blockIdx.x * TPB + threadIdx.x;
    if (i >= npp) return;
    int ntp = (n * m * (n * m - 1) * (n * m - 2)) / 6;
    int x1 = ppx1[i], y1 = ppy1[i], x2 = ppx2[i], y2 = ppy2[i];
    int x_tmp, y_tmp, hash, hash_small1, hash_small2;
    bool found = true;

    int M = (n - 1) + (m - 1) + 1;
    int hash_size = 5 * M;
    // bool* hash_table1 = new bool[hash_size];
    // bool* hash_table2 = new bool[hash_size];
    bool hash_table1[1000];
    bool hash_table2[1000];
    int* distances = new int[n * m];
    int a, b, c;

    for (int j = 0; j < ntp; j++) {
        for (int k = 0; k < hash_size; k++) {
            hash_table1[k] = false;
            hash_table2[k] = false;
        }
        bool flag = true;
        for (int t = 0; t < n * m; t++) {
            x_tmp = px[t];
            y_tmp = py[t];
            a = distf(x_tmp, y_tmp, tpx1[j], tpy1[j], x1, y1, x2, y2);
            b = distf(x_tmp, y_tmp, tpx2[j], tpy2[j], x1, y1, x2, y2);
            c = distf(x_tmp, y_tmp, tpx3[j], tpy3[j], x1, y1, x2, y2);
            hash = a * M * M + b * M + c;
            hash_small1 = (5 * a) ^ (3 * b) ^ (1 * c);
            hash_small2 = (1 * a) ^ (3 * b) ^ (7 * c);
            if (hash_table1[hash_small1] && hash_table2[hash_small2]) {
                for (int k = 0; k < t; k++) {
                    if (distances[k] == hash) {
                        flag = false;
                        break;
                    }
                }
                if (!flag) break;
            } else {
                hash_table1[hash_small1] = true;
                hash_table2[hash_small2] = true;
            }
            distances[t] = hash;
        }
        if (flag) {
            found = false;
            break;
        }
    }
    out[i] = found;
    return;
}

int main(int argc, char* argv[]) {
    if (argc < 3) {
        cout << "Please provide dimensions of the grid.\n";
        exit(-1);
    }
    int n = stoi(argv[1]);
    int m = stoi(argv[2]);

    int points_x[n * m], points_y[n * m];
    for (int i = 1; i <= n; i++) {
        for (int j = 1; j <= m; j++) {
            points_x[m * (i - 1) + j - 1] = i;
            points_y[m * (i - 1) + j - 1] = j;
        }
    }

    vector<int> ppx1, ppy1, ppx2, ppy2;
    for (int i = 0; i < n * m; i++) {
        for (int j = i + 1; j < n * m; j++) {
            int x1 = points_x[i], y1 = points_y[i], x2 = points_x[j],
                y2 = points_y[j];
            if (dist(x1, y1, x2, y2) > 1 && (x1 <= x2) && (y1 >= y2) &&
                (y1 - y2) >= (x2 - x1)) {
                ppx1.push_back(x1);
                ppy1.push_back(y1);
                ppx2.push_back(x2);
                ppy2.push_back(y2);
            }
        }
    }

    int num_pairs = ppx1.size();
    bool conds[num_pairs];
    bool* cond_dev;
    int *ppx1_dev, *ppx2_dev, *ppy1_dev, *ppy2_dev;
    int cond_size = num_pairs * sizeof(bool);
    int pp_size = num_pairs * sizeof(int);
    int num_blocks = ceil((float)num_pairs / TPB);

    hipMalloc((void**)&cond_dev, cond_size);
    hipMalloc((void**)&ppx1_dev, pp_size);
    hipMalloc((void**)&ppy1_dev, pp_size);
    hipMalloc((void**)&ppx2_dev, pp_size);
    hipMalloc((void**)&ppy2_dev, pp_size);

    hipMemcpy(ppx1_dev, &ppx1[0], pp_size, hipMemcpyHostToDevice);
    hipMemcpy(ppy1_dev, &ppy1[0], pp_size, hipMemcpyHostToDevice);
    hipMemcpy(ppx2_dev, &ppx2[0], pp_size, hipMemcpyHostToDevice);
    hipMemcpy(ppy2_dev, &ppy2[0], pp_size, hipMemcpyHostToDevice);

    four_cond<<<num_blocks, TPB>>>(ppx1_dev, ppy1_dev, ppx2_dev, ppy2_dev, n, m,
                                   num_pairs, cond_dev);

    hipMemcpy(conds, cond_dev, cond_size, hipMemcpyDeviceToHost);

    int *tpx1_dev, *tpx2_dev, *tpx3_dev, *tpy1_dev, *tpy2_dev, *tpy3_dev;
    int num_tp = (n * m * (n * m - 1) * (n * m - 2)) / 6;
    int tp_size = num_tp * sizeof(int);

    bool founds[num_pairs];
    bool* founds_dev;
    int p_size = n * m * sizeof(int);
    int *px_dev, *py_dev;

    hipMalloc((void**)&founds_dev, cond_size);
    hipMalloc((void**)&px_dev, p_size);
    hipMalloc((void**)&py_dev, p_size);
    hipMemcpy(px_dev, points_x, p_size, hipMemcpyHostToDevice);
    hipMemcpy(py_dev, points_y, p_size, hipMemcpyHostToDevice);

    hipMalloc((void**)&tpx1_dev, tp_size);
    hipMalloc((void**)&tpx2_dev, tp_size);
    hipMalloc((void**)&tpx3_dev, tp_size);
    hipMalloc((void**)&tpy1_dev, tp_size);
    hipMalloc((void**)&tpy2_dev, tp_size);
    hipMalloc((void**)&tpy3_dev, tp_size);

    make_three_points<<<1, 1>>>(px_dev, py_dev, tpx1_dev, tpy1_dev, tpx2_dev,
                                tpy2_dev, tpx3_dev, tpy3_dev, n, m);

    find<<<num_blocks, TPB>>>(ppx1_dev, ppy1_dev, ppx2_dev, ppy2_dev, tpx1_dev,
                              tpy1_dev, tpx2_dev, tpy2_dev, tpx3_dev, tpy3_dev,
                              px_dev, py_dev, n, m, num_pairs, founds_dev);

    hipMemcpy(founds, founds_dev, cond_size, hipMemcpyDeviceToHost);

    for (int i = 0; i < num_pairs; i++) {
        if (!(conds[i] ^ founds[i])) {
            if (founds[i]) {
                int x1 = ppx1[i], y1 = ppy1[i], x2 = ppx2[i], y2 = ppy2[i];
                cout << "MD is 4 when edge is between (" << x1 << "," << y1
                     << ") and (" << x2 << "," << y2 << ")\n";
            }
        } else {
            cout << "Mistake\n";
            exit(-1);
        }
    }
    cout << "Success!\n";
}